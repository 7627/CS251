
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

struct num_array{
                    double num1;
                    double num2;
                    double result;
};



__device__ void function(struct num_array *a)
{
    double square = a ->num1 * a->num1 +  a->num2 * a->num2  + 2 * a->num1 * a->num2;
    a->result = log(square)/sin(square);
    printf("%f %f %f\n",a->num1,a->num2,a->result);
    return;
}
__global__ void calculate(char *mem, int num)
{
      int i = blockDim.x * blockDim.y * blockIdx.x + (blockDim.y * threadIdx.x) + threadIdx.y; //changed
      printf("blockDim.x=%d .y=%d blockIdx=%d threadIdx.x=%d threadIdx.y=%d\n",blockDim.x,blockDim.y, blockIdx.x , threadIdx.x,threadIdx.y);
      printf("i= %d\n",i);
      if(i >= num)
           return;
       struct num_array *a = (struct num_array *)(mem + (i * 3 * sizeof(double)));
      function(a);
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i;
    struct num_array *pa;
    char *ptr;
    char *sptr;
    char *gpu_mem;
    unsigned long num = atoi(argv[1]);   /*Default value of num from MACRO*/
    int blocks;
    int rows,cols;
    rows=atoi(argv[2]);
    cols=atoi(argv[3]);
    dim3 blockDim(rows,cols);


    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * 3 * sizeof(double));
    sptr = ptr;
    for(i=0; i<num; ++i){
       pa = (struct num_array *) sptr;
       pa->num1 = (double) i + (double) i * 0.1;
       pa->num2 = pa->num1 + 1.0;
       sptr += 3 * sizeof(double);
    }


    gettimeofday(&t_start, NULL);

    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * 3 * sizeof(double));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * 3 * sizeof(double) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    gettimeofday(&start, NULL);

    blocks = num /(rows*cols);

    if(num % (rows*cols))
           ++blocks;

    calculate<<<blocks, blockDim>>>(gpu_mem, num);
    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);

    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * 3 * sizeof(double) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);

    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;

    /*Print the last element for sanity check*/
    pa = (struct num_array *) (sptr + (num -1)*3*sizeof(double));
    printf("num1=%f num2=%f result=%f\n", pa->num1, pa->num2, pa->result);


    free(ptr);
}
