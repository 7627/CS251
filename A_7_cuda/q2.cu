
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>


#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void reduce(int* input, int n){
  unsigned int tid=threadIdx.x;
  unsigned int i=blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int offset = blockDim.x * blockIdx.x;
  __syncthreads();
    for(unsigned int s=1; s<blockDim.x; s*=2){
      if(tid % (2*s)==0){
        input[i] = (i + s < offset + blockDim.x && i+s < n)? input[i] ^ input[i + s] : input[i];
        printf("input[%d]= %d offset= %ld\n",i,input[i],offset);
      }
      __syncthreads();
    }
}

int main(int argc, char** argv){
  struct timeval start, end, t_start, t_end;
  int i,n = atoi(argv[1]);
  int seed= atoi(argv[2]);
  int * array;
  int blocks;
  int result=0;
  int threads=10;
  array=(int*)malloc(n * sizeof(int));

  srand(seed);
  for(i=0;i<n;i++){
    array[i]=random();
    printf("a[%d]= %d\n",i,array[i]);
  }

  int *gpu_array;

  gettimeofday(&t_start, NULL);

  hipMalloc(&gpu_array, n*sizeof(int));
  CUDA_ERROR_EXIT("hipMalloc");

  hipMemcpy(gpu_array, array, n*sizeof(int), hipMemcpyHostToDevice);
  CUDA_ERROR_EXIT("hipMemcpy");

  gettimeofday(&start, NULL);

  blocks= (n + threads -1)/threads;

  reduce<<<blocks,threads>>>(gpu_array,n);
  CUDA_ERROR_EXIT("kernel invocation");
  gettimeofday(&end, NULL);

  hipMemcpy(array, gpu_array, n*sizeof(int), hipMemcpyDeviceToHost);
  CUDA_ERROR_EXIT("memcpy");

  for(i=0;i<n;i+=threads){
    result = result ^ array[i];
    printf("result= %d\n",result);
  }
  gettimeofday(&t_end, NULL);
  printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
  hipFree(gpu_array);

}
